//============================================================================
// Name        : main.cu
// Author      : efti-nile
// Version     :
// Copyright   : WTFPL
// Description : 
//============================================================================

#define SIZE 1000


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <math.h>
#include <omp.h>

using namespace std;

__global__ void func(double *d_A, double *d_B, double *d_C){
	int i = threadIdx.x;
	double di = (double) i;
	d_A[i] = di * sin((di / (double)SIZE) * (double)M_PI);
	d_B[i] = di / (double)SIZE / (double)SIZE + (double)SIZE * (double)SIZE / (di + 1);
	d_C[i] = d_A[i] * d_B[i];
}

int main(int argc, char *argv[]) {
	double *d_A, *d_B, *d_C, *A, *B, *C;

	A = new double[SIZE];
	B = new double[SIZE];
	C = new double[SIZE];

	hipMalloc(&d_A, SIZE * sizeof(double));
	hipMalloc(&d_B, SIZE * sizeof(double));
	hipMalloc(&d_C, SIZE * sizeof(double));

	if(   A == NULL ||   B == NULL ||   C == NULL || \
		d_A == NULL || d_B == NULL || d_C == NULL){
		cout << "Problem allocating memory" << endl;
		if(A != NULL) delete A;
		if(B != NULL) delete B;
		if(C != NULL) delete C;
		if(d_A != NULL) hipFree(d_A);
		if(d_B != NULL) hipFree(d_B);
		if(d_C != NULL) hipFree(d_C);
		exit(1);
	}

	cout << "Allocated total in VRAM and RAM: " << \
			sizeof(double) * 3 * 2 * SIZE / (1024.0 * 1024.0) << " MiB" << endl;

	clock_t begin = clock();

	func<<< 1, SIZE >>>(d_A, d_B, d_C);

	clock_t exec_time = clock() - begin;

	hipMemcpy(A, d_A, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, SIZE * sizeof(double), hipMemcpyDeviceToHost);

	cout << "First 4 elements:" << endl;
	for(int i = 0; i < 4 && i < SIZE; i++){
		cout << fixed \
			 << "A[" << i << "] = " << A[i] \
			 <<	"   B[" << i << "] = " << B[i] \
			 << "   C[" << i << "] = " << C[i] << endl;
	}

	cout << "Executed for N = " << SIZE << " in " << (exec_time * 1000 / CLOCKS_PER_SEC) << " ms" << endl;

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete A;
	delete B;
	delete C;

	return 0;
}
